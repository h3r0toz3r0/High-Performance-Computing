/**
  Question 2
  Compile: nvcc q2.cu -o q2
  Run: srun --pty --nodes 1 --job-name=interactive --partition=gpu --reservation=EECE5640 --gres=gpu:1 ./q2

    DEBUG:
        0   Print histogram
        1   Print file lines to ensure readability of file
        2   Print summation of coauthors per author
        3   Print max co-authors with respective author and execution time

**/

//      Include libraries       //
#include <stdio.h> 
#include <stdlib.h> 
#include <math.h> 
#include <hip/hip_runtime.h>

//      Macros              //
#define DEBUG       3

//      CUDA Kernel             //
__global__ void sort(int *a, int *c, int n){     
    //  Get global threadID
    int id = blockIdx.x*blockDim.x+threadIdx.x; 

    //  Ensure we are within bounds   
    if (id < n){
        if(id == 0){
            c[0] = 0;
        }
        else{
            if(a[id] == a[id - 1]){
                c[id] = 1;
            }
            else{
                c[id] = 0;
            }
        }
    }
} 

//      Main Function           //   
int main( int argc, char* argv[] ){     
    //  Initialize local variables
    char c, string[100];
    int i;
    int N, count, buffer;
    float milliseconds = 0, milliseconds2 = 0;

    //  Timing using CUDA Events
    hipEvent_t start, start2, midpoint, end;
    hipEventCreate(&start);
    hipEventCreate(&start2);
    hipEventCreate(&midpoint);
    hipEventCreate(&end);

    //  Start Clock
    hipEventRecord(start);
    hipEventRecord(start2);

    //  Allocate space
    count = 0;
    buffer = 5;
    int *chunk1 = (int *) malloc (buffer * sizeof(int));
    int *chunk2 = (int *) malloc (buffer * sizeof(int));

    //  Initiate file variables
    FILE *file;
    i = 0;

    //  Open file
    file = fopen("dblp-co-authors.txt","r");

    //  Iterate through each character of the file
    for(c = getc(file); c != EOF; c = getc(file)){
        //  If character is a space
        if (c == ' '){
            //  Initialize to 0
            chunk1[count] = 0;

            //  Convert string to int and place into array
            chunk1[count] = atoi(string);

            //  Replace string with 0s
            memset(string, 0, strlen(string)); 
        }
        //  If character is an end of line
        else if(c == '\n'){
            //  Initialize to 0
            chunk2[count] = 0;

            //  Convert string to int and place into array
            chunk2[count] = atoi(string);

            //  DEBUG Mode 2 ONLY
            if(DEBUG == 1){
                printf("%d %d\n", chunk1[count], chunk2[count]);
            }

            //  Replace string with 0s
            memset(string, 0, strlen(string)); 

            //  Increment counter
            count++;

            //  Reallocate chunk array to ensure it is big enough 
            chunk1 = (int *) realloc(chunk1, sizeof(int) * (count + buffer));
            chunk2 = (int *) realloc(chunk2, sizeof(int) * (count + buffer));
        }

        //  If character is a information we want
        else{
            //  Add character to string array
            strncat(string, &c, 1); 
        }
    }

    //  Initialize zero element
    chunk1[0] = 0;
    chunk2[0] = 2;

    //  Ensure last value gets placed into arrays
    chunk1[317079] = 317079;
    chunk2[317079] = 317080;

    //  Close file
    fclose(file);

    //  Stop midpoint clock
    hipEventRecord(midpoint);
    hipEventElapsedTime(&milliseconds, start2, midpoint);

    //  Distribute data to GPU threads
    N = count;

    //  Allocate space for counting arrays
    int *d_chunk1 = (int *) malloc (N * sizeof(int));
    int *d_count1 = (int *) malloc ((N + 10) * sizeof(int));
    int *count_array1 = (int *) malloc ((N + 10) * sizeof(int));
    int *opposite_count_array = (int *) malloc (N * sizeof(int));

    //  Allocate memory for each vector on GPU
    hipMalloc( &d_chunk1, (N + 1) * sizeof(int) );
    hipMalloc( &d_count1, (N + 1) * sizeof(int) );

    //  Copy vectors to device
    hipMemcpy( d_chunk1, chunk1, N * sizeof(int), hipMemcpyHostToDevice );

    //  Initialize count arrays
    for(i = 0; i < N; i++){
        count_array1[i] = 0;
        opposite_count_array[i] = 0;
    }

    //  Copy vectors to device
    hipMemcpy( d_count1, count_array1, N * sizeof(int), hipMemcpyHostToDevice );

    //  Initialize variables for CUDA
    int blockSize, gridSize;
    blockSize = 1024;                                   //  Number of threads in each thread block
    gridSize = (int) ceil((float) N / blockSize);       //  Number of thread blocks in grid

    //  Execute the kernel
    sort<<<gridSize, blockSize>>>(d_chunk1, d_count1, N * sizeof(int));

    //  Copy back to host
    hipMemcpy( count_array1, d_count1, N * sizeof(int), hipMemcpyDeviceToHost );

    //  Count authors and place into opposite counting array
    int index = 0;
    for(i = 0; i < N; i++){
        if(count_array1[i] == 0){
            opposite_count_array[chunk1[index]] = 1;
        }
        else{
            opposite_count_array[chunk1[index]]++;
        }
        index++;
    }

    //  Count authors in chunk 2
    for(i = 0; i < N; i++){
        opposite_count_array[chunk2[i]]++;
    }

    //  End clock
    hipEventRecord(end);
    hipEventElapsedTime(&milliseconds2, start, end);

    //  Print results
    if(DEBUG == 2){
        printf("Histogram of authors:\n");
        for(i = 1; i < (N / 2); i++){
            if(opposite_count_array[i] != 0){
                printf("(%d, %d)\n", i, opposite_count_array[i]);
            }
        }
        printf("\n");
    }
    else if(DEBUG == 3){
        int max = 0;
        int max_index[N];
        for(i = 1; i < (N / 2); i++){
            if(opposite_count_array[i] != 0){
                if(opposite_count_array[i] >= max){
                    max = opposite_count_array[i];
                }
                max_index[i] = 0;
            }
        }
        for(i = 0; i < (N / 2); i++){
            if(opposite_count_array[i] == max)
                max_index[i] = i;
        }
        printf("\n\n Midpoint Execution Time (milliseconds): %10.8f\n", milliseconds);
        printf("Total Execution Time (milliseconds): %10.8f", milliseconds2);
        printf("\n\nThe largest number of co-authors is %d\n", max);
        printf("The author(s) is/are: ");
        for(i = 0; i < (N / 2); i++){
            if(max_index[i] != 0){
                printf("%d ", i);
            }
        }
        printf("\n\n");
    }
    else if (DEBUG == 0){
        for(i = 1; i < (N / 2); i++){
            count_array1[i] = 0;
            if(opposite_count_array[i] != 0){
                count_array1[opposite_count_array[i]]++;
            }
        }
        for(i = 0; i < (N / 2); i++){
            //  Author ID, co-author count
            if(count_array1[i] != 0)
            printf("%d,%d\n", i, count_array1[i]);
        }
    }

    //  Cleanly exit
    hipFree(d_chunk1);
    hipFree(d_count1);
    free(chunk1);
    free(chunk2);
    free(count_array1);
    free(opposite_count_array);

    return 0; 
} 
 
